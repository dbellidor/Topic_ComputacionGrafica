#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "MedianFilter.h"
#include <time.h>

__global__ void kernel_filtro(unsigned char *inputImageKernel, unsigned char *outputImagekernel, int imageWidth, int imageHeight)
{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned char filterVector[9] = {0,0,0,0,0,0,0,0,0};   //Longitud de la ventana
	if((row==0) || (col==0) || (row==imageHeight-1) || (col==imageWidth-1))
				outputImagekernel[row*imageWidth+col] = 0; //Condiciones limite
	else {
		for (int x = 0; x < WINDOW_SIZE; x++) { 
			for (int y = 0; y < WINDOW_SIZE; y++){
				filterVector[x*WINDOW_SIZE+y] = inputImageKernel[(row+x-1)*imageWidth+(col+y-1)];   // llenamos el vector de la mediana
			}
		}
		for (int i = 0; i < 9; i++) {
			for (int j = i + 1; j < 9; j++) {
				if (filterVector[i] > filterVector[j]) { 
					//Ordenamiento
					char tmp = filterVector[i];
					filterVector[i] = filterVector[j];
					filterVector[j] = tmp;
				}
			}
		}
		outputImagekernel[row*imageWidth+col] = filterVector[4];   //Guardamos el elemento de la mediana
	}
}


__global__ void CMF_kernel(unsigned char* in, unsigned char * out, int c) {
	// k = input array index
	int k = c+blockDim.x*blockIdx.x+threadIdx.x;
	int M[WINDOW_SIZE] = {0}; //major array
	int e[WINDOW_SIZE] = {0}; //equal array
	int m[WINDOW_SIZE] = {0}; //minor array
	//window index
	int i = k-c+threadIdx.y;
	for (int j=k-c ; j<k-c+WINDOW_SIZE ; j++) {
		if (in[j] > in[i]) M[threadIdx.y]+=1;
		else if (in[j] < in[i]) m[threadIdx.y]+=1;
		else e[threadIdx.y]+=1;
	}
	for (int j = 0; j<WINDOW_SIZE; j++) {
		if (M[j]==c || m[j]==c || e[j]>=c) {
		out[k] = in[k-c+j];
		return;
		}
	}
}


bool GPU_filtro_mediana( Bitmap* image, Bitmap* outputImage, bool sharedMemoryUse ){
	//Eventos de cuda para manejar errores
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	int width = image->Width();
	int height = image->Height();
	int TILE_SIZE=4;
	int size =  width * height * sizeof(char);
	//Inicializamos las imagenes
	unsigned char *deviceinputimage;
	hipMalloc((void**) &deviceinputimage, size);

	hipMemcpy(deviceinputimage, image->image, size, hipMemcpyHostToDevice);

	unsigned char *deviceOutputImage;
	hipMalloc((void**) &deviceOutputImage, size);
	//Tamanio del bloque y del grid
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float)image->Width() / (float)TILE_SIZE),(int)ceil((float)image->Height() / (float)TILE_SIZE));
	
	kernel_filtro<<<dimGrid, dimBlock>>>(deviceinputimage, deviceOutputImage, width, height);
	hipEventRecord(stop);
	//Invocacion de la funcion kernel
	// Guardamos la salida en el host
	hipMemcpy(outputImage->image, deviceOutputImage, size, hipMemcpyDeviceToHost);

	//Liberar memoria
	hipFree(deviceinputimage);
	hipFree(deviceOutputImage);
	float time = 0;
	hipEventElapsedTime(&time,start,stop);
	printf("Tiempo %f\n", 100.0*(time/1000)); 
	return true;
}

bool CMF_GPU( Bitmap* image, Bitmap* outputImage){

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	int width = image->Width();
	int height = image->Height();
	int nthread=512;
	
	int size =  width * height * sizeof(char);
	
	int bkd=(nthread-(nthread % WINDOW_SIZE)) / WINDOW_SIZE;
	int grd=((size-(size % bkd))/bkd)+1;
	dim3 thdims(bkd, WINDOW_SIZE, 1);
	dim3 bkdims(grd, 1, 1);

	//Inicializar imagenes
	unsigned char *deviceinputimage;
	hipMalloc((void**) &deviceinputimage, size);
	hipMemcpy(deviceinputimage, image->image, size, hipMemcpyHostToDevice);
	
	unsigned char *deviceOutputImage;
	hipMalloc((void**) &deviceOutputImage, size);

	CMF_kernel<<<bkdims,thdims>>>(deviceinputimage, deviceOutputImage, 2);
	hipEventRecord(stop);
	// Guardar imagen en host
	hipMemcpy(outputImage->image, deviceOutputImage, size, hipMemcpyDeviceToHost);
	
	//Liberar memoria
	hipFree(deviceinputimage);
	hipFree(deviceOutputImage);
	float time = 0;
	hipEventElapsedTime(&time,start,stop);
	printf("Tiempo %f\n", 100.0*((time)/1000)); 
	return true;
}

