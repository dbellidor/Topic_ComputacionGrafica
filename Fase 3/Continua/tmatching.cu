#include "hip/hip_runtime.h"
// OpenCV library
#include <opencv/cv.h>
#include <opencv/highgui.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <list>
#include <time.h>
using namespace std;
using namespace cv;
#define VALUE_MAX 10000000

#define CPU

struct match {

	int diffRow;
	int diffCol;
	int diff;

}pos;
list<int> listx;
list<int> listy;
int contador=0;

// Funcion CPU

void GPU_ImgMatching(IplImage* sourceImg, IplImage* patternImg, int value);
// Funcion GPU
void CPU_matching(IplImage* sourceImg, IplImage* patternImg, int value);

__global__ void GPU_Kernel_ImgMatching(unsigned char * d_ImgSrc ,unsigned char *d_pImgSrc,int *d_diffDst,int w1,int h1, int w2, int h2,int value);

int main(int argc, char *argv[]){

	IplImage* sourceImg; 
	IplImage* patternImg;
	
  	
	sourceImg = cvLoadImage("Raspberry.png",0);
	patternImg = cvLoadImage("r1.png",0);

	GPU_ImgMatching(sourceImg,patternImg,83);
	CPU_matching(sourceImg,patternImg,83);
	list<int>::iterator itx;
	list<int>::iterator ity;
	CvPoint pt1, pt2;
	IplImage* original=cvLoadImage("Raspberry.png");
	for(itx=listx.begin(),ity=listy.begin();itx!=listx.end(),ity!=listy.end();itx++,ity++){
		//cout<<"x "<<*itx<<" y "<<*ity ;
		pt1.x = *itx;
		pt1.y = *ity;
		pt2.x = pt1.x + patternImg->width;
		pt2.y = pt1.y + patternImg->height;
		cvRectangle(original,cvPoint(pt1.x,pt1.y),cvPoint(pt2.x,pt2.y),CV_RGB(255,0,0),1,8);
	
	}
	cvSaveImage("match.png", original);


	return 0;

}

void CPU_matching(IplImage* sourceImg, IplImage* patternImg, int value){
	clock_t start, end;
	double cpu_time_used;
	start = clock();

	int w1=sourceImg->width;
	int h1=sourceImg->height;
	int w2=patternImg->width;
	int h2=patternImg->height;
	int R1= (w2*h2) - ((w2*h2*value)/100);
	int R2=0;
	int w=w1-w2;
	int h=h1-h2;
	int x1,y1;
	int x2,y2;
	bool bandera;
	uchar color1, color2;
	  for(y1=0 ; y1 < h ;y1++){
		for( x1=0 ; x1 < w ; x1++){
		    bandera=true;
		    for(y2=0 ; y2 < h2 && bandera ; y2++){
		        for(x2=0 ; x2 < w2 && bandera ; x2++){
			    color1 = sourceImg->imageData[(y1 + y2) * sourceImg->widthStep+x1+x2];
			    color2 = patternImg->imageData[y2 * patternImg->widthStep+x2];
		            if(abs(color2 - color1) >25){
		                R2++;
		                if(R2>R1){
		                    bandera=false;
		                }
		            }
		        }
		    }
		    if(R2<=R1){
		        if(listx.empty() && listy.empty()){
		            listx.push_back(x1);
			    listy.push_back(y1);
		            contador++;
		            
		        }else if(x1>listx.back()+w2/2 || y1>listy.back()+h2/2){
		            listx.push_back(x1);
			    listy.push_back(y1);
		            contador++;
		            
		        }
		    }
		    R2=0;
		}
    	  }

	end = clock();
	cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("Tiempo CPU %lf\n", 100.0*cpu_time_used); 
}

__global__ void GPU_Kernel_ImgMatching(unsigned char * d_ImgSrc ,unsigned char *d_pImgSrc,int *d_diffDst,int w1,int h1, int w2, int h2,int value)
{
	
	int R1= (w2*h2) - ((w2*h2*value)/100);
	int R2=0;
        int w=w1-w2;
	int h=h1-h2;
	bool bandera;
	int x1,y1;
	int x2,y2;
	uchar color1, color2;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int contador=0;
	if(tid < w1 ) {
		for(y1=0 ; y1 < h ;y1++){
			for( x1=0 ; x1 < w ; x1++){
			    bandera=true;
			    for(y2=0 ; y2 < h2 && bandera ; y2++){
				for(x2=0 ; x2 < w2 && bandera ; x2++){
				    color1 = d_ImgSrc[(y1 + y2) * w1 +  tid +x2];
				    color2 = d_pImgSrc[y2 * w2 + x2];
				    if(abs(color2 - color1) >25){
				        R2++;
				        if(R2>R1){
				            bandera=false;
				        }
				    }
				}
			    }
			    if(R2<=R1){
				if(d_diffDst==NULL){
					d_diffDst[contador]=x1;
					contador++;
				}
			    }
			    R2=0;
			}
	    	  }
	}


}

void GPU_ImgMatching(IplImage* sourceImg, IplImage* patternImg, int value)
{
	
	int w1=sourceImg->width;
	int h1=sourceImg->height;
	int w2=patternImg->width;
	int h2=patternImg->height;

	// GPU Size
	//size_t d_sizeDiff = sizeof(int) * (w1 - w2 + 1) * (h1 - h2 + 1) ;
	size_t d_sizeDiff = sizeof(int) * 10;
	size_t d_sizeImg  = sizeof(unsigned char) * w1 * h1;
	size_t d_psizeImg = sizeof(unsigned char) * w2 * h2;


	// CPU
	unsigned char *h_ImgSrc  = (unsigned char*)(sourceImg->imageData);
	unsigned char *h_pImgSrc = (unsigned char*)(patternImg->imageData);
	int *h_diffDst = (int *)malloc(d_sizeDiff);

	// GPU 
	int *d_diffDst = NULL;
	unsigned char *d_ImgSrc  = NULL;
	unsigned char *d_pImgSrc = NULL;


	// GPU reserva de memoria
	hipMalloc((void**)&d_diffDst, d_sizeDiff);
	hipMalloc((void**)&d_ImgSrc , d_sizeImg);
	hipMalloc((void**)&d_pImgSrc, d_psizeImg);


	//Copia de host a device
	hipMemcpy(d_diffDst, h_diffDst, d_sizeDiff, hipMemcpyHostToDevice);
	hipMemcpy(d_pImgSrc, h_pImgSrc, d_psizeImg, hipMemcpyHostToDevice);
	hipMemcpy(d_ImgSrc , h_ImgSrc , d_sizeImg , hipMemcpyHostToDevice);

	// Definir el bloque y los hilos
    	dim3 dimGrid(8);
   	dim3 dimBlock(128);
	// Cuda time profile
    	hipEvent_t start, stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);
    	hipEventRecord(start, 0);

	// the kernel function
    	GPU_Kernel_ImgMatching<<<dimGrid, dimBlock>>>(d_ImgSrc, d_pImgSrc, d_diffDst, w1, h1, w2, h2,value);
	// Manejar el error de la función Kernel.
    	hipError_t cuda_err = hipGetLastError();
    	if ( hipSuccess != cuda_err ){
        	printf("Antes de la llamada al kernel: error = %s\n", hipGetErrorString (cuda_err));
        	exit(1) ;
    	}
	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);

    	float elapsedTime;
    	hipEventElapsedTime(&elapsedTime, start, stop);
    	printf("Tiempo GPU :\t%13f seg\n", elapsedTime/1000);
	hipEventDestroy(start);
    	hipEventDestroy(stop);
	//Copiar memoria de device a host  
    	hipMemcpy(h_diffDst, d_diffDst, d_sizeDiff, hipMemcpyDeviceToHost);
}




